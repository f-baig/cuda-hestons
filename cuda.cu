#include "hip/hip_runtime.h"
#include "MonteCarloSimulator.h"
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <iostream>

__global__ void monteCarloKernel(double* d_results, double S, double K, double T, double r, double sigma, int num_simulations) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < num_simulations) {
        hiprandState state;
        hiprand_init(1234, idx, 0, &state);

        double gauss_bm = hiprand_normal(&state);

        double ST = S * exp((r - 0.5 * sigma * sigma) * T + sigma * sqrt(T) * gauss_bm);

        // Call
        d_results[idx] = max(ST - K, 0.0); 
    }
}

void MonteCarloSimulator::launchCUDASimulation(double* d_results) {
    int threads_per_block = 256;
    int number_of_blocks = (num_simulations + threads_per_block - 1) / threads_per_block;

    monteCarloKernel<<<number_of_blocks, threads_per_block>>>(d_results, S, K, T, r, sigma, num_simulations);

    hipDeviceSynchronize();
}

double MonteCarloSimulator::runSimulationCUDA() {
    double* d_results;
    double* h_results = new double[num_simulations];

    hipMalloc((void**)&d_results, num_simulations * sizeof(double));

    launchCUDASimulation(d_results);

    hipMemcpy(h_results, d_results, num_simulations * sizeof(double), hipMemcpyDeviceToHost);

    double payoff_sum = 0.0;
    for (int i = 0; i < num_simulations; ++i) {
        payoff_sum += h_results[i];
    }

    hipFree(d_results);
    delete[] h_results;

    return (payoff_sum / static_cast<double>(num_simulations)) * exp(-r * T);
}
